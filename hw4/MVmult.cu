// $ nvcc -arch=sm_61 MVmult.cu -o MVmult -Xcompiler -fopenmp
// flag -Xcompiler passes next flag directly to compiler

#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>
#include <random>


// we will assume that m and n are divisible by THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 1024
#define m (THREADS_PER_BLOCK * 10)
#define n (THREADS_PER_BLOCK * 5)

void MVmult(double* b, const double* A, const double* x) {
  #pragma omp parallel for schedule(static)
  for ( long i = 0; i < m; i++ ) {
    double sum = 0;
    for ( long j = 0; j < n; j++ ) {
      sum += A[i*n + j] * x[j];
    }
    b[i] = sum;
  }
}


__global__ 
void MVmult_kernel(double* b, const double* A, const double* x) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  double sum = 0.0;
  if ( idx < m ) {
    for ( long i = 0; i < n; i++ ) {
      sum += A[idx*n + i] * x[i];
    }
    b[idx] = sum;
  }
}


// second idea where all products are calculated in parallel, this ended up being slower.
__global__ 
void MVmult_kernel2(double* b, const double* A, const double* x) {
  __shared__ double prods[THREADS_PER_BLOCK]; //shared var for all products
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int j = idx%n; //column
  int i = idx / n; //row
  prods[threadIdx.x] = A[idx]*x[j];
  __syncthreads(); //sync to make sure all relavent prods are calculated
  if (0 == threadIdx.x ) {
    double sum = 0;
    for ( int k = 0; k < THREADS_PER_BLOCK; k++ ) {
      sum += prods[k];
    }
    atomicAdd(&b[i],sum);
  }
}


void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}


int main(void) {
  // notation: Ax = b with A m by n
  double* A = (double*) malloc( m * n * sizeof(double));
  double* x = (double*) malloc( n * sizeof(double));
  double* b = (double*) malloc( m * sizeof(double));
  double* b_ref = (double*) malloc( m * sizeof(double));
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < m*n; i++) {
    A[i]   = rand() % 10 -10;
  }
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < n; i++) {
    x[i]   = rand() % 10 -10;
  }

  double tt = omp_get_wtime();
  MVmult(b_ref, A, x);
  printf("CPU %f s\n", omp_get_wtime()-tt);

  double *A_d, *x_d, *b_d;
  hipMalloc(&A_d, m * n *sizeof(double));
  Check_CUDA_Error("malloc x failed");
  hipMalloc(&x_d, n *sizeof(double));
  hipMalloc(&b_d, m *sizeof(double));

  tt = omp_get_wtime();
  hipMemcpy(A_d, A, m * n *sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(x_d, x, n *sizeof(double), hipMemcpyHostToDevice);

  double ttinner = omp_get_wtime();
  MVmult_kernel<<< m/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(b_d, A_d, x_d);
  // MVmult_kernel2<<<n*m/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(b_d, A_d, x_d);
  hipDeviceSynchronize();
  ttinner = omp_get_wtime() - ttinner;
  hipMemcpy(b, b_d, m *sizeof(double), hipMemcpyDeviceToHost);
  printf("GPU %f s, %f s\n", omp_get_wtime()-tt, ttinner);

  double err = 0;
  for (long i = 0; i < m; i++) err += fabs(b[i]-b_ref[i]);
  printf("Error = %f\n", err);

  //printf("b[0] = %f\n", b[0]);

  hipFree(A_d);
  hipFree(x_d);
  hipFree(b_d);

  free(A);
  free(x);
  free(b);
  free(b_ref);

  return 0;
}
